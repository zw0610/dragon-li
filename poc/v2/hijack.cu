#include <hip/hip_runtime.h>

#include <stdio.h>
#include <dlfcn.h>
#include <stdlib.h>

hipError_t hipDeviceTotalMem(size_t* bytes, hipDevice_t dev) {
    void *handle;
    handle = dlopen("/usr/lib/x86_64-linux-gnu/libcuda.so.1", RTLD_LAZY);

    printf("%s\n", "hipDeviceTotalMem is hijacked based on env MYMEM!");

    const char* mymem = getenv("MYMEM");
    int env_set_mem_i = 0;
    sscanf(mymem, "%d", &env_set_mem_i);
    size_t env_set_mem = env_set_mem_i < 0 ? 0 : env_set_mem_i;

    hipError_t (*ori_cu_device_total_mem)(size_t*, hipDevice_t);
    ori_cu_device_total_mem = (hipError_t (*)(size_t *, hipDevice_t))dlsym(handle, "hipDeviceTotalMem");

    size_t total_mem = 0;
    hipError_t res = ori_cu_device_total_mem(&total_mem, dev);

    *bytes = env_set_mem < total_mem ? env_set_mem : total_mem;

    dlclose(handle);
    return res;
}
