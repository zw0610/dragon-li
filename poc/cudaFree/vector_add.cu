
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 10;
 
    // Device input vectors
    double *d_a;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
    
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);

    // Free memory 
    hipFree(d_a);
 
    return 0;
}
