#include <hip/hip_runtime.h>

#include <stdio.h>
#include <dlfcn.h>
#include <stdlib.h>

hipError_t hipDeviceTotalMem(size_t* bytes, hipDevice_t dev) {
    void *handle;
    handle = dlopen("/usr/lib/x86_64-linux-gnu/libcuda.so.1", RTLD_LAZY);

    printf("%s\n", "I just want to tell you that hipDeviceTotalMem is STILL hijacked!");
    
    hipError_t (*ori_cu_device_total_mem)(size_t*, hipDevice_t);
    ori_cu_device_total_mem = (hipError_t (*)(size_t *, hipDevice_t))dlsym(handle, "hipDeviceTotalMem");

    hipError_t res = ori_cu_device_total_mem(bytes, dev);

    dlclose(handle);
    return res;
}
