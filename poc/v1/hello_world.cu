#include <hip/hip_runtime.h>

#include <iostream>


#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr,
                "CUDA Driver API error = %04d from file <%s>, line %i.\n",
                err, file, line );
        exit(-1);
    }
}

int main() {

    checkCudaErrors(hipInit(0));

    int device_count = 0;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    std::cout << "Hello World!" << std::endl;
    std::cout << "Find " << device_count << " device(s)." << std::endl;

    hipDevice_t device;
    checkCudaErrors(hipDeviceGet(&device, 0));

    char name[100];
    checkCudaErrors(hipDeviceGetName(name, 100, device));

    size_t bytes = 0;
    checkCudaErrors(hipDeviceTotalMem(&bytes, device));

    std::cout << name << " has total memory of " << bytes << " bytes." << std::endl;

    return 0;
}
